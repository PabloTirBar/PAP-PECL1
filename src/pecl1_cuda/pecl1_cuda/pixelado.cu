#include "hip/hip_runtime.h"
#include "pixelado.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernelPixelar(byte* d_pixels, int width, int height, int bytesPerPixel, int blockSize) {
    int bx = blockIdx.x * blockSize;
    int by = blockIdx.y * blockSize;

    if (bx >= width || by >= height) return;

    int sumR = 0, sumG = 0, sumB = 0;
    int count = 0;

    for (int i = 0; i < blockSize && (by + i) < height; ++i) {
        for (int j = 0; j < blockSize && (bx + j) < width; ++j) {
            int idx = ((by + i) * width + (bx + j)) * bytesPerPixel;
            sumR += d_pixels[idx + 2];
            sumG += d_pixels[idx + 1];
            sumB += d_pixels[idx + 0];
            count++;
        }
    }

    byte avgR = sumR / count;
    byte avgG = sumG / count;
    byte avgB = sumB / count;

    for (int i = 0; i < blockSize && (by + i) < height; ++i) {
        for (int j = 0; j < blockSize && (bx + j) < width; ++j) {
            int idx = ((by + i) * width + (bx + j)) * bytesPerPixel;
            d_pixels[idx + 0] = avgB;
            d_pixels[idx + 1] = avgG;
            d_pixels[idx + 2] = avgR;
        }
    }
}

void pixelarImagen(byte* h_pixels, int width, int height, int bytesPerPixel, int blockSize) {
    byte* d_pixels;
    size_t size = width * height * bytesPerPixel;

    hipMalloc(&d_pixels, size);
    hipMemcpy(d_pixels, h_pixels, size, hipMemcpyHostToDevice);

    dim3 gridSize((width + blockSize - 1) / blockSize, (height + blockSize - 1) / blockSize);

    kernelPixelar << <gridSize, 1 >> > (d_pixels, width, height, bytesPerPixel, blockSize);
    hipDeviceSynchronize();

    hipMemcpy(h_pixels, d_pixels, size, hipMemcpyDeviceToHost);
    hipFree(d_pixels);
}