#include <iostream>
#include <string>
#include <cstdlib>
#include "utils.h"
#include "blanco_negro.cuh"
#include "pixelado.cuh"
#include "identificar_colores.cuh"
#include "filtrar_delinear.cuh"
#include "pseudo_hash.cuh"

using namespace std;

void mostrarMenu() {
    cout << "\n========= MEN� DE OPCIONES =========" << endl;
    cout << "0. Salir" << endl;
    cout << "1. Crear copia de la imagen base" << endl;
    cout << "2. Conversi�n a blanco y negro" << endl;
    cout << "3. Pixelar imagen" << endl;
    cout << "4. Identificaci�n de colores" << endl;
    cout << "5. Filtrado y delineado de zonas de color" << endl;
    cout << "6. C�lculo de pseudo-hash" << endl;
    cout << "====================================" << endl;
    cout << "Seleccione una opci�n: ";
}

int main() {
    string rutaImagen;
    cout << "Introduce la ruta de la imagen BMP (ENTER para usar la ruta por defecto): ";
    getline(cin, rutaImagen);
    if (rutaImagen.empty()) {
        rutaImagen = "C:\\Users\\iubal.camjalli\\Documents\\PAP\\PAP-PECL1\\imgs\\input.bmp";
    }

    // Variables comunes para todas las fases
    byte* pixels;
    int32 ancho, alto, bytesPerPixel;

    if (!cargarBMP(rutaImagen.c_str(), &pixels, &ancho, &alto, &bytesPerPixel)) {
        cerr << "No se pudo cargar la imagen base. Terminando programa." << endl;
        return -1;
    }

    int opcion;
    do {
        mostrarMenu();
        cin >> opcion;
        cin.ignore(); // Limpiar el buffer

        switch (opcion) {
        case 1: {
            string rutaSalida;
            cout << "Ruta para guardar la copia: ";
            getline(cin, rutaSalida);
            if (!exportarBMP(rutaSalida.c_str(), pixels, ancho, alto, bytesPerPixel)) {
                cerr << "No se pudo guardar la copia de la imagen." << endl;
            }
            else {
                cout << "Imagen copiada con �xito en: " << rutaSalida << endl;
            }
            break;
        }
        case 2: {
            convertirBlancoNegro(pixels, ancho, alto, bytesPerPixel);
            string rutaSalida = "C:\\Users\\iubal.camjalli\\Documents\\PAP\\PAP-PECL1\\imgs\\blanco_y_negro.bmp";
            if (!exportarBMP(rutaSalida.c_str(), pixels, ancho, alto, bytesPerPixel)) {
                cerr << "Error al guardar la imagen blanco y negro." << endl;
            }
            else {
                cout << "Imagen en blanco y negro guardada en: " << rutaSalida << endl;
            }
            break;
        }
        case 3: {
            pixelarImagen(pixels, ancho, alto, bytesPerPixel, 8); // Tama�o de bloque 8 como ejemplo
            string rutaSalida = "C:\\Users\\iubal.camjalli\\Documents\\PAP\\PAP-PECL1\\imgs\\pixelado.bmp";
            if (!exportarBMP(rutaSalida.c_str(), pixels, ancho, alto, bytesPerPixel)) {
                cerr << "Error al guardar la imagen pixelada." << endl;
            }
            else {
                cout << "Imagen pixelada guardada en: " << rutaSalida << endl;
            }
            break;
        }
        case 4: {
            float umbral, magnitud;

            cout << "Introduce el umbral (valor por defecto 30): ";
            string input;
            getline(cin, input);
            umbral = input.empty() ? 30.0f : std::stof(input);

            cout << "Introduce el factor de magnitud (valor por defecto 1.0): ";
            getline(cin, input);
            magnitud = input.empty() ? 1.0f : std::stof(input);

            byte* copiaRojo = (byte*)malloc(ancho * alto * bytesPerPixel);
            byte* copiaVerde = (byte*)malloc(ancho * alto * bytesPerPixel);
            byte* copiaAzul = (byte*)malloc(ancho * alto * bytesPerPixel);
            memcpy(copiaRojo, pixels, ancho * alto * bytesPerPixel);
            memcpy(copiaVerde, pixels, ancho * alto * bytesPerPixel);
            memcpy(copiaAzul, pixels, ancho * alto * bytesPerPixel);

            identificarColor(copiaRojo, ancho, alto, bytesPerPixel, ROJO, umbral, magnitud,
                "C:\\Users\\iubal.camjalli\\Documents\\PAP\\PAP-PECL1\\imgs\\identificar_rojo.bmp");

            identificarColor(copiaVerde, ancho, alto, bytesPerPixel, VERDE, umbral, magnitud,
                "C:\\Users\\iubal.camjalli\\Documents\\PAP\\PAP-PECL1\\imgs\\identificar_verde.bmp");

            identificarColor(copiaAzul, ancho, alto, bytesPerPixel, AZUL, umbral, magnitud,
                "C:\\Users\\iubal.camjalli\\Documents\\PAP\\PAP-PECL1\\imgs\\identificar_azul.bmp");

            free(copiaRojo);
            free(copiaVerde);
            free(copiaAzul);
            break;
        }
        case 5: {
            float umbral, magnitud;

            cout << "Introduce el umbral (valor por defecto 30): ";
            string input;
            getline(cin, input);
            umbral = input.empty() ? 30.0f : std::stof(input);

            cout << "Introduce el factor de magnitud (valor por defecto 1.0): ";
            getline(cin, input);
            magnitud = input.empty() ? 1.0f : std::stof(input);

            byte* copiaRojo = (byte*)malloc(ancho * alto * bytesPerPixel);
            byte* copiaVerde = (byte*)malloc(ancho * alto * bytesPerPixel);
            byte* copiaAzul = (byte*)malloc(ancho * alto * bytesPerPixel);
            memcpy(copiaRojo, pixels, ancho * alto * bytesPerPixel);
            memcpy(copiaVerde, pixels, ancho * alto * bytesPerPixel);
            memcpy(copiaAzul, pixels, ancho * alto * bytesPerPixel);

            filtrarYDelimitarColor(copiaRojo, ancho, alto, bytesPerPixel, "rojo", umbral, magnitud);
            filtrarYDelimitarColor(copiaVerde, ancho, alto, bytesPerPixel, "verde", umbral, magnitud);
            filtrarYDelimitarColor(copiaAzul, ancho, alto, bytesPerPixel, "azul", umbral, magnitud);

            free(copiaRojo);
            free(copiaVerde);
            free(copiaAzul);
            break;
        }
        case 6: {
            mostrarPseudoHash(pixels, ancho, alto, bytesPerPixel);
            break;
        }
        case 0:
            cout << "Saliendo del programa..." << endl;
            break;
        default:
            cout << "Opci�n no v�lida. Intente de nuevo." << endl;
        }

    } while (opcion != 0);

    free(pixels);
    return 0;
}